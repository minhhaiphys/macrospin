#include "hip/hip_runtime.h"
#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __shared__
#endif

#include <stdio.h>
#include <pycuda-complex.hpp>

// ============================================================
//                      Helper Functions
// ============================================================
inline __device__ float4 cross(float4 left, float4 right) {
  return make_float4(left.y*right.z - left.z*right.y,
		     left.z*right.x - left.x*right.z,
		     left.x*right.y - left.y*right.x,
		     0.0f);
}
inline __device__ float4 operator+(float4 a, float4 b) {
  return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, 0.0f);
}
inline __device__ float4 mult(float4 a, float4 b) {
  return make_float4(a.x*b.x, a.y*b.y, a.z*b.z, 0.0f);
}
inline __device__ float4 operator-(float4 a, float4 b) {
  return make_float4(a.x-b.x, a.y-b.y, a.z-b.z, 0.0f);
}
inline __device__ float4 operator*(float b, float4 a) {
  return make_float4(a.x*b, a.y*b, a.z*b, 0.0f);
}
inline __device__ float magInv(float4 a) {
  return rsqrtf(a.x*a.x + a.y*a.y + a.z*a.z);
}

// ============================================================
//                      Physics Code
// ============================================================

__global__ void evolve(float4 *m) {

  {{ definitions }}
  {{ index_operations }}

  {{ static_field }}
  {{ demagnetization }}
  {{ uniaxial_anisotropy }}
  {{ cubic_anisotropy }}

  // Start Landau-Lifshitz equation
  float4 hxm = cross(heff, mloc);

  {{ spin_transfer_torque }}

  float4 mxhxm =  cross(mloc, hxm);

  // Compute new moment
  m[i] = mloc + {{ dt }}*(hxm + {{ damping }}*mxhxm);
}

__global__ void normalize(float4 *m) {
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  float4 mloc = m[i];
  m[i] = magInv(mloc)*mloc;
}